#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <torch/torch.h>

#include <cmath>
#include <vector>

// hashing
// input N*4 int32 tensor output N*1 int64 tensor
__global__ void hash_kernel(int N, const int *__restrict__ data,
                            int64_t *__restrict__ out) {
  int i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < N) {
    data += i * 4;
    uint64_t hash = 14695981039346656037UL;
    for (int j = 0; j < 4; j++) {
      hash ^= (unsigned int)data[j];
      hash *= 1099511628211UL;
    }
    hash = (hash >> 60) ^ (hash & 0xFFFFFFFFFFFFFFF);
    out[i] = hash;
  }
}

// kernel hashing: given data D and offset map K, generate D x K
// input N*4 int32 tensor, |K|*3 int32 tensor, output |K|*N int64 tensor
__global__ void kernel_hash_kernel(int N, int K, const int *__restrict__ data,
                                   const int *__restrict__ kernel_offset,
                                   int64_t *__restrict__ out) {
  extern __shared__ int kernel_offset_local[];

  for (int i = 0; i < K * 3; i++) {
    kernel_offset_local[i] = kernel_offset[i];
  }
  __syncthreads();

  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  int k = idx % K;
  int i = idx / K;
  int cur_coord[4];
  if (i < N) {
    data += i * 4;
    for (int j = 0; j < 3; j++) {
      cur_coord[j] = data[j] + kernel_offset[k * 3 + j];
    }
    cur_coord[3] = data[3];
    uint64_t hash = 14695981039346656037UL;
    for (int j = 0; j < 4; j++) {
      hash ^= (unsigned int)cur_coord[j];
      hash *= 1099511628211UL;
    }
    hash = (hash >> 60) ^ (hash & 0xFFFFFFFFFFFFFFF);
    out[k * N + i] = hash;
  }
}

void kernel_hash_wrapper(int N, int K, const int *data,
                         const int *kernel_offset, int64_t *out) {
  kernel_hash_kernel<<<ceil((double)(N * K) / 512), 512, K * 3 * sizeof(int)>>>(
      N, K, data, kernel_offset, out);
}

void hash_wrapper(int N, const int *data, int64_t *out) {
  hash_kernel<<<ceil((double)N / 512), 512>>>(N, data, out);
}

at::Tensor hash_cuda(const at::Tensor idx) {
  int N = idx.size(0);
  at::Tensor out =
      torch::zeros({N}, at::device(idx.device()).dtype(at::ScalarType::Long));
  hash_wrapper(N, idx.data_ptr<int>(), out.data_ptr<int64_t>());
  return out;
}

at::Tensor kernel_hash_cuda(const at::Tensor idx,
                            const at::Tensor kernel_offset) {
  int N = idx.size(0);
  int K = kernel_offset.size(0);
  at::Tensor out = torch::zeros(
      {K, N}, at::device(idx.device()).dtype(at::ScalarType::Long));
  kernel_hash_wrapper(N, K, idx.data_ptr<int>(), kernel_offset.data_ptr<int>(),
                      out.data_ptr<int64_t>());
  return out;
}
