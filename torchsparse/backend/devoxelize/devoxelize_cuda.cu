#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <thrust/device_vector.h>
#include <torch/extension.h>

#include <THC/THCAtomics.cuh>

// input features (n, c), indices (N, 8), weight (N, 8) -> output features (N,
// c)
template <typename scalar_t>
__global__ void devoxelize_forward_kernel(int N, int c,
                                          const int *__restrict__ indices,
                                          const scalar_t *__restrict__ weight,
                                          const scalar_t *__restrict__ feat,
                                          scalar_t *__restrict__ out) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int i = index / c;
  int j = index % c;

  if (i < N) {
    const int *indices_ = indices + 8 * i;
    const scalar_t *weight_ = weight + 8 * i;
    const scalar_t *feat_ = feat + j;

    scalar_t cur_feat;
    for (int k = 0; k < 8; k++) {
      cur_feat = 0;
      if (indices_[k] >= 0) cur_feat = feat_[indices_[k] * c];

      out[i * c + j] += weight_[k] * cur_feat;
    }
  }
}

// input weight (N, 8), indices (N, 8), top_grad (N, c) -> bottom grad (n, c)
template <typename scalar_t>
__global__ void devoxelize_backward_kernel(
    int N, int n, int c, const int *__restrict__ indices,
    const scalar_t *__restrict__ weight, const scalar_t *__restrict__ top_grad,
    scalar_t *__restrict__ bottom_grad) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int i = index / c;
  int j = index % c;

  if (i < N) {
    const int *indices_ = indices + 8 * i;
    const scalar_t *weight_ = weight + 8 * i;

    scalar_t cur_top_grad = top_grad[i * c + j];

#pragma unroll
    for (int k = 0; k < 8; k++) {
      if (indices_[k] >= 0)
        atomicAdd(&bottom_grad[indices_[k] * c + j], weight_[k] * cur_top_grad);
    }
  }
}

// make sure indices is int type
// feat: (b,c,s) indices: (N, 3) batch_index: (N, ) -> out: (N, c)
at::Tensor devoxelize_forward_cuda(const at::Tensor feat,
                                   const at::Tensor indices,
                                   const at::Tensor weight) {
  int c = feat.size(1);
  int N = indices.size(0);

  at::Tensor out =
      torch::zeros({N, c}, at::device(feat.device()).dtype(feat.dtype()));

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      feat.type(), "devoxelize_forward_cuda", ([&] {
        devoxelize_forward_kernel<scalar_t><<<N, c>>>(
            N, c, indices.data_ptr<int>(), weight.data_ptr<scalar_t>(),
            feat.data_ptr<scalar_t>(), out.data_ptr<scalar_t>());
      }));

  return out;
}

// top_grad: (N, c), indices: (N, 3), batch_index: (N, ) -> bottom_grad:
// (b,c,s), s=r^3
at::Tensor devoxelize_backward_cuda(const at::Tensor top_grad,
                                    const at::Tensor indices,
                                    const at::Tensor weight, int n) {
  int c = top_grad.size(1);
  int N = top_grad.size(0);
  at::Tensor bottom_grad = torch::zeros(
      {n, c}, at::device(top_grad.device()).dtype(top_grad.dtype()));

  AT_DISPATCH_FLOATING_TYPES_AND_HALF(
      top_grad.type(), "devoxelize_backward_cuda", ([&] {
        devoxelize_backward_kernel<scalar_t><<<N, c>>>(
            N, n, c, indices.data_ptr<int>(), weight.data_ptr<scalar_t>(),
            top_grad.data_ptr<scalar_t>(), bottom_grad.data_ptr<scalar_t>());
      }));

  return bottom_grad;
}
